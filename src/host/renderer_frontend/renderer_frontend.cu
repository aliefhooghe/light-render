#include "hip/hip_runtime.h"

#include <math.h>

#include "gpu/common/abstract_image_developer.cuh"
#include "gpu/common/gpu_texture.cuh"
#include "gpu/common/renderer_manager.cuh"
#include "gpu/image_developers/average_image_developer.cuh"
#include "gpu/image_developers/gamma_image_developer.cuh"
#include "gpu/model/bvh_tree.cuh"
#include "gpu/model/camera.cuh"
#include "gpu/renderers/naive_mc_renderer.cuh"
#include "gpu/renderers/preview_renderer.cuh"
#include "gpu/utils/cuda_exception.cuh"
#include "gpu/utils/device_probing.cuh"
#include "gpu/utils/gpu_vector_copy.cuh"
#include "host/bvh_builder/bvh_builder.cuh"
#include "host/camera_handling/camera_configuration.cuh"
#include "host/camera_handling/camera_configuration.cuh"
#include "host/model_loader/wavefront_obj.cuh"
#include "host/utils/chronometer.h"
#include "renderer_frontend.h"

namespace Xrender
{
    class renderer_frontend_implementation
    {

    public:
        using setting = renderer_frontend::setting;
        using worker_descriptor = renderer_frontend::worker_descriptor;

        renderer_frontend_implementation(
            camera cam,
            const host_bvh_tree::gpu_compatible_bvh &gpu_bvh,
            GLuint texture_id);
        ~renderer_frontend_implementation() noexcept;

        void scale_sensor_lens_distance(bool up, float factor);
        void scale_focal_length(bool up, float factor);
        void scale_diaphragm_radius(bool up, float factor);
        void camera_move(float dx, float dy, float dz);
        void camera_move_forward(float distance);
        void camera_move_lateral(float distance);
        void camera_rotate(float theta, float phi);

        void integrate_for(const std::chrono::milliseconds &max_duration);
        void develop_image();
        std::vector<rgb24> get_image();
        std::size_t get_renderer_count() const;
        void set_current_renderer(std::size_t renderer_id);
        std::size_t get_current_renderer() const;
        const worker_descriptor &get_renderer_descriptor(std::size_t renderer_id) const;

        std::size_t get_developer_count() const;
        void set_current_developer(std::size_t developer_id);
        std::size_t get_current_developer() const;
        const worker_descriptor &get_developer_descriptor(std::size_t developer_id) const;

        unsigned int get_image_width() const noexcept { return _camera.get_image_width(); }
        unsigned int get_image_height() const noexcept { return _camera.get_image_height(); }

    private:
        void _add_renderer(
            worker_descriptor&& descriptor,
            std::unique_ptr<abstract_renderer> &&renderer);
        void _add_image_developer(
            worker_descriptor&& descriptor,
            std::unique_ptr<abstract_image_developer> &&developer);
        void _reset_current_renderer();

        camera _camera;
        bvh_node *_device_tree{nullptr};
        face *_device_model{nullptr};

        std::size_t _current_renderer{0u};
        std::size_t _current_developer{0u};

        std::vector<renderer_manager> _renderers{};
        std::vector<std::unique_ptr<abstract_image_developer>> _developpers{};

        std::vector<worker_descriptor> _renderers_settings{};
        std::vector<worker_descriptor> _developpers_settings{};

        registered_texture _registered_texture;
    };

    /**
     * Private implementation
     */

    renderer_frontend_implementation::renderer_frontend_implementation(
        camera cam,
        const host_bvh_tree::gpu_compatible_bvh &gpu_bvh,
        GLuint texture_id)
    :   _camera{cam},
        _registered_texture{texture_id, cam.get_image_width(), cam.get_image_height()}
    {
        _device_tree = clone_to_device(gpu_bvh.tree);
        _device_model = clone_to_device(gpu_bvh.model);

        // Add average developer
        {
            auto average_developer = std::make_unique<average_image_developer>();
            auto *average_dev = average_developer.get();

            _add_image_developer(
                {
                    "Average Developer",
                    {
                        {
                            "Factor", [average_dev](bool up) { average_dev->scale_factor(up); }
                        }
                    }
                },
                std::move(average_developer));
        }

        // Add gamma developer
        {
            auto gamma_developer = std::make_unique<gamma_image_developer>();
            auto *gamma_dev = gamma_developer.get();

            _add_image_developer(
                {
                    "Gamma developer",
                    {
                        {
                            "Factor", [gamma_dev](bool up) { gamma_dev->scale_factor(up); }
                        },
                        {
                            "Gamma",  [gamma_dev](bool up) { gamma_dev->scale_gamma(up); }
                        }
                    }
                },
                std::move(gamma_developer));
        }

        _add_renderer(
            {"Preview", {}},
            std::make_unique<preview_renderer>(_device_tree, _device_model));

        _add_renderer(
            {"Path Tracer", {}},
            std::make_unique<naive_mc_renderer>(_device_tree, _device_model));
    }

    renderer_frontend_implementation::~renderer_frontend_implementation() noexcept
    {
        CUDA_WARNING(hipFree(_device_tree));
        CUDA_WARNING(hipFree(_device_model));
    }

    void renderer_frontend_implementation::_add_renderer(
        worker_descriptor&& descriptor,
        std::unique_ptr<abstract_renderer> &&renderer)
    {
        _renderers_settings.emplace_back(std::move(descriptor));
        _renderers.emplace_back(_camera, std::move(renderer));
    }

    void renderer_frontend_implementation::_add_image_developer(
        worker_descriptor&& descriptor,
        std::unique_ptr<abstract_image_developer> &&developer)
    {
        _developpers_settings.emplace_back(std::move(descriptor));
        _developpers.emplace_back(std::move(developer));
    }

    void renderer_frontend_implementation::_reset_current_renderer()
    {
        if (_current_renderer >= get_renderer_count())
            return;
        _renderers[_current_renderer].reset();
    }
    void renderer_frontend_implementation::scale_sensor_lens_distance(bool up, float factor)
    {
        if (up)
            _camera._sensor_lens_distance *= factor;
        else
            _camera._sensor_lens_distance /= factor;
        _reset_current_renderer();
    }

    void renderer_frontend_implementation::scale_focal_length(bool up, float factor)
    {
        camera_update_focal_length(_camera, up, factor);
        _reset_current_renderer();
    }

    void renderer_frontend_implementation::scale_diaphragm_radius(bool up, float factor)
    {
        if (up)
            _camera._diaphragm_radius *= factor;
        else
            _camera._diaphragm_radius /= factor;
        _reset_current_renderer();
    }

    void renderer_frontend_implementation::camera_move(float dx, float dy, float dz)
    {
        _camera._position += float3{dx, dy, dz};
        _reset_current_renderer();
    }

    void renderer_frontend_implementation::camera_move_forward(float distance)
    {
        camera_update_pos_forward(_camera, distance);
        _reset_current_renderer();
    }

    void renderer_frontend_implementation::camera_move_lateral(float distance)
    {
        camera_update_pos_lateral(_camera, distance);
        _reset_current_renderer();
    }

    void renderer_frontend_implementation::camera_rotate(float theta, float phi)
    {
        camera_update_rotation(_camera, theta, phi);
        _reset_current_renderer();
    }



    void renderer_frontend_implementation::integrate_for(const std::chrono::milliseconds &max_duration)
    {
        if (_current_renderer >= get_renderer_count())
            return;
        _renderers[_current_renderer].integrate_for(max_duration);
    }

    void renderer_frontend_implementation::develop_image()
    {
        if (_current_developer >= get_developer_count() || _current_renderer >= get_renderer_count())
            return;

        auto mapped_surface = _registered_texture.get_mapped_surface();
        auto &renderer = _renderers[_current_renderer];

        _developpers[_current_developer]->call_develop_to_texture_kernel(
            renderer.get_total_sample_count(),
            _camera.get_image_width(),
            _camera.get_image_height(),
            renderer.get_device_sensor(),
            mapped_surface.surface());

        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    }

    std::vector<rgb24> renderer_frontend_implementation::get_image()
    {
        const auto host_texture = _registered_texture.retrieve_texture();
        std::vector<rgb24> image{host_texture.size()};

        // Convert to 24 bit bitmap samples
        std::transform(
            host_texture.begin(), host_texture.end(),
            image.begin(),
            [](const float4& rgba)
            {
                return rgb24::from_float(rgba.x, rgba.y, rgba.z);
            });

        return image;
    }

    std::size_t renderer_frontend_implementation::get_renderer_count() const
    {
        return _renderers.size();
    }

    void renderer_frontend_implementation::set_current_renderer(std::size_t renderer_id)
    {
        if (renderer_id < get_renderer_count()) {
            _current_renderer = renderer_id;
            _reset_current_renderer();
        }
        else {
            throw std::invalid_argument("invalid renderer id");
        }
    }

    std::size_t renderer_frontend_implementation::get_current_renderer() const
    {
        return _current_renderer;
    }

    const renderer_frontend_implementation::worker_descriptor &renderer_frontend_implementation::get_renderer_descriptor(std::size_t renderer_id) const
    {
        if (renderer_id < get_renderer_count())
            return _renderers_settings[renderer_id];
        else
            throw std::invalid_argument("invalid renderer id");
    }

    std::size_t renderer_frontend_implementation::get_developer_count() const
    {
        return _developpers.size();
    }

    void renderer_frontend_implementation::set_current_developer(std::size_t developer_id)
    {
        if (developer_id < get_developer_count()) {
            _current_developer = developer_id;
        }
        else {
            throw std::invalid_argument("invalid renderer id");
        }
    }

    std::size_t renderer_frontend_implementation::get_current_developer() const
    {
        return _current_developer;
    }

    const renderer_frontend_implementation::worker_descriptor &renderer_frontend_implementation::get_developer_descriptor(std::size_t developer_id) const
    {
        if (developer_id < get_renderer_count())
            return _developpers_settings[developer_id];
        else
            throw std::invalid_argument("invalid renderer id");
    }

    /**
     * Private implementation wrapping
     */

    std::unique_ptr<renderer_frontend> renderer_frontend::build_renderer_frontend(const render_configuration &configuration, GLuint texture_id)
    {
        if (!select_openGL_cuda_device())
            throw std::runtime_error("No cuda capable device was found");

        // Load model
        chronometer timewatch{};

        std::cout << "Loading " << configuration.model_path.generic_string() << std::endl;
        timewatch.start();
        const auto model = wavefront_obj_load(configuration.model_path);
        const auto load_duration = timewatch.stop();

        // Create a bvh usable on gpu
        std::cout << "Model loading took " << load_duration.count() << " ms\nBuild bvh tree (" << model.size() << " faces)" << std::endl;
        timewatch.start();
        const auto host_bvh = build_bvh_tree(model);
        const auto gpu_bvh = host_bvh->to_gpu_bvh();
        const auto bvh_build_duration = timewatch.stop();
        std::cout << "Bvh build took " << bvh_build_duration.count() << " ms" << std::endl;
        std::cout << "Bvh tree max depth is " << host_bvh->max_depth() << std::endl;
        std::cout << "GPU bvh tree size is  : " << gpu_bvh.tree.size() << std::endl;
        std::cout << "GPU bvh model size is : " << gpu_bvh.model.size() << std::endl;

        // Configure camera
        camera cam{};
        configure_camera(configuration.camera_config, cam);

        // Initialize the frontend implementation
        std::cout << "Initialize computations" << std::endl;
        auto *implementation =
            new renderer_frontend_implementation{cam, gpu_bvh, texture_id};

        return std::make_unique<renderer_frontend>(implementation);
    }

    renderer_frontend::renderer_frontend(renderer_frontend&& other) noexcept
    :   _implementation{other._implementation}
    {
        other._implementation = nullptr;
    }

    renderer_frontend::renderer_frontend(renderer_frontend_implementation *implementation)
    :   _implementation{implementation}
    {
    }

    renderer_frontend::~renderer_frontend() noexcept
    {
        if (_implementation)
            delete _implementation;
    }

    void renderer_frontend::scale_sensor_lens_distance(bool up, float factor)
    {
        _implementation->scale_sensor_lens_distance(up, factor);
    }

    void renderer_frontend::scale_focal_length(bool up, float factor)
    {
        _implementation->scale_focal_length(up, factor);
    }

    void renderer_frontend::scale_diaphragm_radius(bool up, float factor)
    {
        _implementation->scale_diaphragm_radius(up, factor);
    }

    void renderer_frontend::camera_move(float dx, float dy, float dz)
    {
        _implementation->camera_move(dx, dy, dz);
    }

    void renderer_frontend::camera_move_forward(float distance)
    {
        _implementation->camera_move_forward(distance);
    }

    void renderer_frontend::camera_move_lateral(float distance)
    {
        _implementation->camera_move_lateral(distance);
    }

    void renderer_frontend::camera_rotate(float theta, float phi)
    {
        _implementation->camera_rotate(theta, phi);
    }

    void renderer_frontend::integrate_for(const std::chrono::milliseconds &max_duration)
    {
        _implementation->integrate_for(max_duration);
    }

    void renderer_frontend::develop_image()
    {
        _implementation->develop_image();
    }

    std::vector<rgb24> renderer_frontend::get_image()
    {
        return _implementation->get_image();
    }

    unsigned int renderer_frontend::get_image_width() const noexcept
    {
        return _implementation->get_image_width();
    }

    unsigned int renderer_frontend::get_image_height() const noexcept
    {
        return _implementation->get_image_height();
    }


    std::size_t renderer_frontend::get_renderer_count() const
    {
        return _implementation->get_renderer_count();
    }

    void renderer_frontend::set_current_renderer(std::size_t renderer_id)
    {
        _implementation->set_current_renderer(renderer_id);
    }

    std::size_t renderer_frontend::get_current_renderer() const
    {
        return _implementation->get_current_renderer();
    }

    const renderer_frontend::worker_descriptor &renderer_frontend::get_renderer_descriptor(std::size_t renderer_id) const
    {
        return _implementation->get_renderer_descriptor(renderer_id);
    }

    std::size_t renderer_frontend::get_developer_count() const
    {
        return _implementation->get_developer_count();
    }

    void renderer_frontend::set_current_developer(std::size_t developer_id)
    {
        _implementation->set_current_developer(developer_id);
    }

    std::size_t renderer_frontend::get_current_developer() const
    {
        return _implementation->get_current_developer();
    }

    const renderer_frontend::worker_descriptor &renderer_frontend::get_developer_descriptor(std::size_t developer_id) const
    {
        return _implementation->get_developer_descriptor(developer_id);
    }

}