#include "hip/hip_runtime.h"

#include <algorithm>
#include <iostream>
#include <thread>
#include <iostream>

#include "random_generator.cuh"
#include "bvh_builder.cuh"

namespace Xrender
{

    /**
     * \brief return half the surface of a aabb box
     */
    static __host__ float aabb_box_half_area(const aabb_box &box)
    {
        const auto lengths = box.ext_max - box.ext_min;
        return (lengths.x * lengths.y +
                lengths.x * lengths.z +
                lengths.y * lengths.z);
    }

    /**
     * \brief build the smallest aabb box containing all given faces
     * \param begin iterator to a face pointer container
     */
    template <typename Titerator>
    static __host__ aabb_box make_aabb_box(Titerator begin, Titerator end)
    {
        aabb_box box = {
            {INFINITY, INFINITY, INFINITY},
            {-INFINITY, -INFINITY, -INFINITY}};

        for (auto it = begin; it != end; ++it)
        {
            box.ext_min = min(box.ext_min, min((*it)->geo.points[2], min((*it)->geo.points[1], (*it)->geo.points[0])));
            box.ext_max = max(box.ext_max, max((*it)->geo.points[2], max((*it)->geo.points[1], (*it)->geo.points[0])));
        }

        return box;
    }

    /**
     * \brief compute the coordinate of a face gravity center projected on a line
     * \param f face to be projected
     * \param dir the direction of the line
     */
    static __host__ float face_axis_value(const triangle& f, const float3 &dir)
    {
        return dot(dir, f.points[0] + f.points[1] + f.points[2]);
    }

    /**
     * \brief Compute the variance of the faces gravity centers projected on a line
     */
    template <typename Titerator>
    static __host__ float axis_variance(const float3 &dir, Titerator begin, Titerator end)
    {
        const float count = end - begin;
        float sum = 0.0f;
        float square_sum = 0.0f;

        for (auto it = begin; it != end; ++it)
        {
            const auto value = face_axis_value((*it)->geo, dir);
            sum += value;
            square_sum += (value * value);
        }

        return (square_sum - (sum * sum)) / (count * count);
    }



    /**
     * \brief Sample some random axis directions and return the one
     * with the greatest axis variance
     */
    template <typename Titerator>
    static __host__ float3 find_axis(Titerator begin, Titerator end, std::size_t sample_count)
    {
        float3 best_axis{};
        float best_variance = -INFINITY;

        for (auto i = 0u; i < sample_count; ++i)
        {
            const float3 axis = rand::unit_sphere_uniform();
            const float axis_var = axis_variance(axis, begin, end);

            if (axis_var > best_variance)
            {
                best_variance = axis_var;
                best_axis = axis;
            }
        }

        return best_axis;
    }

    /**
     * \brief compute the SAH heuristic for a given partition if it is better thant the best one
     */
    template <typename Titerator>
    static __host__ float compute_partition_sah_heuristic(Titerator begin, Titerator partition, Titerator end)
    {
        const auto count1 = partition - begin;
        const auto count2 = end - partition;

        const auto box1 = make_aabb_box(begin, partition);
        const auto box2 = make_aabb_box(partition, end);

        return (float)count1 * aabb_box_half_area(box1) + (float)count2 * aabb_box_half_area(box2);
    }

    template <typename Titerator>
    auto partition_by_index(
        std::size_t partition_index,
        Titerator face_begin, Titerator face_end,
        std::size_t total_partition_count)
    {
        const auto face_count = face_end - face_begin;
        const auto face_idx = (partition_index + 1) * face_count / (total_partition_count + 1);
        return face_begin + face_idx;
    }

    template <typename Titerator>
    auto find_min_sah(
        Titerator face_begin, Titerator face_end,
        std::size_t partition_begin, std::size_t partition_end, std::size_t total_partition_count,
        float& sah)
    {
        Titerator best_partition =
            partition_by_index(partition_begin, face_begin, face_end, total_partition_count);
        float best_sah =
            compute_partition_sah_heuristic(face_begin, best_partition, face_end);

        if (partition_end == partition_begin) throw;

        for (auto i = partition_begin + 1; i < partition_end; ++i)
        {
            const auto partition =
                partition_by_index(i, face_begin, face_end, total_partition_count);
            const float partition_sah =
                compute_partition_sah_heuristic(face_begin, partition, face_end);

            if (partition_sah < best_sah)
            {
                best_sah = partition_sah;
                best_partition = partition;
            }
        }

        sah = best_sah;
        return best_partition;
    }

    template <typename Titerator>
    auto parallel_find_min_sah(std::size_t thread_count, Titerator begin, Titerator end)
    {
        constexpr auto min_pivot_per_thread = 12;

        const auto count = end - begin;
        const auto pivot_test_count = std::min(count - 1l, 256l);

        std::size_t current_pivot = 0u;
        const std::size_t blocksize = pivot_test_count / (thread_count + 1u);

        std::vector<std::pair<Titerator, float>> results{thread_count};
        std::vector<std::thread> workers{thread_count};

        if (blocksize >= min_pivot_per_thread)
        {
            for (auto i = 0u; i < thread_count; i++)
            {
                auto worker_begin = current_pivot;
                auto worker_end = current_pivot + blocksize;

                workers[i] = std::thread{
                    [i, &results, begin, end, pivot_test_count](std::size_t pbegin, std::size_t pend)
                    {
                        results[i].first =
                            find_min_sah(
                                begin, end,
                                pbegin, pend, pivot_test_count,
                                results[i].second);
                    },
                    worker_begin, worker_end};

                current_pivot = worker_end;
            }
        }

        float best_sah;
        Titerator best_partition =
            find_min_sah(
                begin, end,
                current_pivot, pivot_test_count, pivot_test_count,
                best_sah);

        if (blocksize >= min_pivot_per_thread)
        {
            for (auto i = 0u; i < thread_count; i++)
            {
                workers[i].join();

                if (results[i].second < best_sah)
                {
                    best_partition = results[i].first;
                    best_sah = results[i].second;
                }
            }
        }

        return best_partition;
    }

    /**
     * \brief Try to find the partition with the lower SAH heuristic
     */
    template <typename Titerator>
    static __host__ auto find_partition(Titerator begin, Titerator end, std::size_t axis_sample_count)
    {
        // Generate an axis along which face are spread
        const auto sort_axis = find_axis(begin, end, axis_sample_count);

        // Sort the faces according to the axis
        std::sort(
            begin, end,
            [&sort_axis](const face *f1, const face *f2) -> bool
            {
                return face_axis_value(f1->geo, sort_axis) >
                       face_axis_value(f2->geo, sort_axis);
            });

        // Find the best partition
        return parallel_find_min_sah(15, begin, end);
    }


    template <typename Titerator>
    static __host__ host_bvh_tree::node build_node(Titerator begin, Titerator end);

    /**
     * \brief Build a branch with the given faces (The must be more than one face)
     */
    template <typename Titerator>
    static __host__ std::unique_ptr<host_bvh_tree> build_branch(Titerator begin, Titerator end)
    {
        // find best partition and build childs
        const auto partition = find_partition(begin, end, 32);

        // Build node
        auto branch = std::make_unique<host_bvh_tree>();
        branch->box = make_aabb_box(begin, end);
        branch->left_child = build_node(begin, partition);
        branch->right_child = build_node(partition, end);

        return branch;
    }

    /**
     * \brief Build a node with the given faces
     */
    template <typename Titerator>
    static __host__ host_bvh_tree::node build_node(Titerator begin, Titerator end)
    {
        // Only one face
        if (begin + 1 == end)
        {
            // return face as leaf
            return *begin;
        }
        else
        {
            return build_branch(begin, end);
        }
    }

    __host__ std::unique_ptr<host_bvh_tree> build_bvh_tree(const std::vector<face>& model)
    {
        const auto face_count = model.size();
        std::vector<const face *> model_faces{face_count};

        // Get faces ptr in a buffer (in order to sort them)
        std::transform(
            model.begin(), model.end(), model_faces.begin(),
            [](const auto& f) { return &f; });

        return build_branch(model_faces.begin(), model_faces.end());
    }

}