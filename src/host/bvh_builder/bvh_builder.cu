#include "hip/hip_runtime.h"

#include <algorithm>
#include <iostream>

#include "random_generator.cuh"
#include "bvh_builder.cuh"

namespace Xrender
{
    /**
     * \brief return half the surface of a aabb box
     */
    static __host__ float aabb_box_half_area(const aabb_box &box)
    {
        const auto lengths = box.ext_max - box.ext_min;
        return (lengths.x * lengths.y +
                lengths.x * lengths.z +
                lengths.y * lengths.z);
    }

    /**
     * \brief build the smallest aabb box containing all given faces
     * \param begin iterator to a face pointer container
     */
    template <typename Titerator>
    static __host__ aabb_box make_aabb_box(Titerator begin, Titerator end)
    {
        aabb_box box = {
            {INFINITY, INFINITY, INFINITY},
            {-INFINITY, -INFINITY, -INFINITY}};

        for (auto it = begin; it != end; ++it)
        {
            box.ext_min = min(box.ext_min, min((*it)->points[2], min((*it)->points[1], (*it)->points[0])));
            box.ext_max = max(box.ext_max, max((*it)->points[2], max((*it)->points[1], (*it)->points[0])));
        }

        return box;
    }

    /**
     * \brief compute the coordinate of a face gravity center projected on a line
     * \param f face to be projected
     * \param dir the direction of the line
     */
    static __host__ float face_axis_value(const face *f, const float3 &dir)
    {
        return dot(dir, f->points[0] + f->points[1] + f->points[2]);
    }

    /**
     * \brief Compute the variance of the faces gravity centers projected on a line
     */
    template <typename Titerator>
    static __host__ float axis_variance(const float3 &dir, Titerator begin, Titerator end)
    {
        const float count = end - begin;
        float sum = 0.0f;
        float square_sum = 0.0f;

        for (auto it = begin; it != end; ++it)
        {
            const auto value = face_axis_value(*it, dir);
            sum += value;
            square_sum += (value * value);
        }

        return (square_sum - (sum * sum)) / (count * count);
    }



    /**
     * \brief Sample some random axis directions and return the one
     * with the greatest axis variance
     */
    template <typename Titerator>
    static __host__ float3 find_axis(Titerator begin, Titerator end, std::size_t sample_count)
    {
        float3 best_axis{};
        float best_variance = -INFINITY;

        for (auto i = 0u; i < sample_count; ++i)
        {
            const float3 axis = rand::unit_sphere_uniform();
            const float axis_var = axis_variance(axis, begin, end);

            if (axis_var > best_variance)
            {
                best_variance = axis_var;
                best_axis = axis;
            }
        }

        return best_axis;
    }

    /**
     * \brief compute the SAH heuristic for a given partition
     */
    template <typename Titerator>
    static __host__ float compute_partition_sah_heuristic(Titerator begin1, Titerator begin2, Titerator end)
    {
        const auto count1 = begin2 - begin1; // end
        const auto count2 = end - begin2;

        const auto box1 = make_aabb_box(begin1, begin2);
        const auto box2 = make_aabb_box(begin2, end);

        return (float)count1 * aabb_box_half_area(box1) + (float)count2 * aabb_box_half_area(box2);
    }

    /**
     * \brief Try to find the partition with the lower SAH heuristic
     */
    template <typename Titerator>
    static __host__ auto find_partition(Titerator begin, Titerator end, std::size_t axis_sample_count)
    {
        const auto count = end - begin;
        const auto pivot_test_count = std::min(count - 1l, 1000l);

        // Generate an axis along which face are spread
        const auto sort_axis = find_axis(begin, end, axis_sample_count);

        // Sort the faces according to the axis
        std::sort(
            begin, end,
            [&sort_axis](const face *f1, const face *f2) -> bool
            {
                return face_axis_value(f1, sort_axis) >
                       face_axis_value(f2, sort_axis);
            });

        // Find the best partition
        float best_sah = INFINITY;                    // less is better
        Titerator best_partition = begin + count / 2; // TODO

        for (auto i = 0; i < pivot_test_count; ++i)
        {
            const auto partition = begin + (i * (count - 1u)) / pivot_test_count;
            const float partition_sah = compute_partition_sah_heuristic(begin, partition, end);

            if (partition_sah < best_sah)
            {
                best_sah = partition_sah;
                best_partition = partition;
            }
        }

        return best_partition;
    }


    template <typename Titerator>
    static __host__ host_bvh_tree::node build_node(Titerator begin, Titerator end);

    /**
     * \brief Build a branch with the given faces (The must be more than one face)
     */
    template <typename Titerator>
    static __host__ std::unique_ptr<host_bvh_tree> build_branch(Titerator begin, Titerator end)
    {
        // Build node bounding box
        auto branch = std::make_unique<host_bvh_tree>();
        branch->box = make_aabb_box(begin, end);

        // find best partition and build childs
        const auto partition = find_partition(begin, end, 32);
        branch->left_child = build_node(begin, partition);
        branch->right_child = build_node(partition, end);

        return branch;
    }

    /**
     * \brief Build a node with the given faces
     */
    template <typename Titerator>
    static __host__ host_bvh_tree::node build_node(Titerator begin, Titerator end)
    {
        // Only one face
        if (begin + 1 == end)
        {
            // return face as leaf
            return **begin;
        }
        else
        {
            return build_branch(begin, end);
        }
    }

    __host__ std::unique_ptr<host_bvh_tree> build_bvh_tree(const std::vector<face>& model)
    {
        const auto face_count = model.size();
        std::vector<const face *> model_faces{face_count};

        // Get faces ptr in a buffer (in order to sort them)
        std::transform(
            model.begin(), model.end(), model_faces.begin(),
            [](const auto& f) { return &f; });

        return build_branch(model_faces.begin(), model_faces.end());
    }

}