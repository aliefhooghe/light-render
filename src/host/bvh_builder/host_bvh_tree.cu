#include "hip/hip_runtime.h"

#include <stdexcept>
#include "host_bvh_tree.cuh"

namespace Xrender
{
    __host__ static void _push_node(std::vector<bvh_node>& gpu_tree, const host_bvh_tree::node& host_node);

    __host__ static void _push_branch(std::vector<bvh_node>& gpu_tree, const host_bvh_tree& branch)
    {
        // Push root
        bvh_node gpu_node;
        gpu_node.type = bvh_node::BOX;
        gpu_node.node.box = branch.box;

        const auto root_index = gpu_tree.size();
        gpu_tree.emplace_back(std::move(gpu_node));

        // Push left child
        _push_node(gpu_tree, branch.left_child);

        // Update root info
        gpu_tree[root_index].node.second_child_idx = gpu_tree.size();

        // Push right child
        _push_node(gpu_tree, branch.right_child);
    }

    __host__ static void _push_child(std::vector<bvh_node>& gpu_tree, const host_bvh_tree::parent& parent)
    {
        _push_branch(gpu_tree, *parent);
    }

    __host__ static void _push_child(std::vector<bvh_node>& gpu_tree, const host_bvh_tree::leaf& leaf)
    {
        bvh_node gpu_leaf;
        gpu_leaf.type = bvh_node::LEAF;
        gpu_leaf.leaf = leaf;
        gpu_tree.emplace_back(std::move(gpu_leaf));
    }

    __host__ static void _push_node(std::vector<bvh_node>& gpu_tree, const host_bvh_tree::node& host_node)
    {
        std::visit(
            [&gpu_tree](auto& child) { _push_child(gpu_tree, child);},
            host_node);
    }

    __host__ std::vector<bvh_node> host_bvh_tree::to_gpu_bvh() const
    {
        if (max_depth() > BVH_MAX_DEPTH)
            throw std::invalid_argument("Bvh depth is too high for gpu");

        std::vector<bvh_node> gpu_tree{};
        _push_branch(gpu_tree, *this);
        gpu_tree.shrink_to_fit();
        return gpu_tree;
    }

    __host__ static std::size_t _child_max_depth(const host_bvh_tree::parent& parent)
    {
        return parent->max_depth();
    }

    __host__ static std::size_t _child_max_depth(const host_bvh_tree::leaf& leaf)
    {
        return 1u;
    }

    __host__ static std::size_t _node_max_depth(const host_bvh_tree::node& host_node)
    {
        return std::visit(
            [](auto& child) { return _child_max_depth(child);},
            host_node);
    }

    __host__ std::size_t host_bvh_tree::max_depth() const noexcept
    {
        return std::max(
            _node_max_depth(left_child),
            _node_max_depth(right_child)
        );
    }

}