#include "hip/hip_runtime.h"

#include "gpu/utils/cuda_exception.cuh"

#include "gpu_texture.cuh"

namespace Xrender
{

    gpu_texture::mapped_surface::mapped_surface(gpu_texture& texture)
    :   _texture{&texture}
    {
        hipArray_t array;

        CUDA_CHECK(hipGraphicsMapResources(1, &(_texture->_graphic_resource), nullptr));
        CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&array, _texture->_graphic_resource, 0, 0));

        hipResourceDesc resource_desc;
        resource_desc.resType = hipResourceTypeArray;
        resource_desc.res.array.array = array;

        CUDA_CHECK(hipCreateSurfaceObject(&_surface_object, &resource_desc));
    }

    gpu_texture::mapped_surface::mapped_surface(gpu_texture::mapped_surface&& other) noexcept
    :   _texture{other._texture},
        _surface_object{other._surface_object}
    {
        other._texture = nullptr;
    }

    __host__ gpu_texture::mapped_surface::~mapped_surface() noexcept
    {
        if (_texture != nullptr) {
            CUDA_WARNING(hipDestroySurfaceObject(_surface_object));
            CUDA_WARNING(hipGraphicsUnmapResources(1, &(_texture->_graphic_resource), NULL));
        }
    }

    __host__ hipSurfaceObject_t gpu_texture::mapped_surface::surface() const noexcept
    {
        return _surface_object;
    }

    __host__ gpu_texture::gpu_texture(unsigned int width, unsigned int height)
        : _width{width}, _height{height}
    {
        glGenTextures(1, &_gl_id);
        glBindTexture(GL_TEXTURE_2D, _gl_id);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, _width, _height, 0, GL_RGBA, GL_FLOAT, NULL);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        glEnable(GL_TEXTURE_2D);

        CUDA_CHECK(hipGetLastError())

        CUDA_CHECK(hipGraphicsGLRegisterImage(
            &_graphic_resource, _gl_id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone));
    }

    __host__ gpu_texture::~gpu_texture() noexcept
    {
        CUDA_WARNING(hipGraphicsUnregisterResource(_graphic_resource));
        glDeleteTextures(1, &_gl_id);
    }

    __host__ unsigned int gpu_texture::get_width() const noexcept
    {
        return _width;
    }

    __host__ unsigned int gpu_texture::get_height() const noexcept
    {
        return _height;
    }

    __host__ GLuint gpu_texture::get_gl_texture_id() const noexcept
    {
        return _gl_id;
    }

    __host__ gpu_texture::mapped_surface gpu_texture::get_mapped_surface()
    {
        return mapped_surface{*this};
    }

    __host__ std::vector<float4> gpu_texture::retrieve_texture()
    {
        std::vector<float4> host_texture{_width * _height};
        hipArray_t array;

        CUDA_CHECK(hipGraphicsMapResources(1, &_graphic_resource, nullptr));
        CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&array, _graphic_resource, 0, 0));

        CUDA_CHECK(hipMemcpy2DFromArray(
            host_texture.data(), _width * sizeof(float4), array,
            0, 0, _width * sizeof(float4), _height, hipMemcpyDeviceToHost));

        CUDA_WARNING(hipGraphicsUnmapResources(1, &_graphic_resource, NULL));
        return host_texture;
    }
}