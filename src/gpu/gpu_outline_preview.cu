#include "hip/hip_runtime.h"

#include <algorithm>
#include <chrono>

#include "gpu_outline_preview.cuh"
#include "vector_operations.cuh"
#include "cuda_exception.cuh"
#include "rand_operations.cuh"

namespace Xrender {


    __global__ void preview_kernel(
        const gpu_face *model, 
        const int face_count,
        const device_camera cam,
        const int sample_count, 
        float3 *image)
    {
        //  Get pixel position in image
        const int x = threadIdx.x;
        const int y = blockIdx.x;
        const int width = blockDim.x;
        const int pixel_index = x + y * width;

        //  Initialize random generator
        hiprandState rand_state;
        hiprand_init(1984+pixel_index, 0, 0, &rand_state);

        float3 pos;
        float3 dir;     
        gpu_intersection inter;
        float3 estimator = {0.f, 0.f, 0.f};

        for (auto i = 0; i < sample_count; i++) {
            dir = cam.sample_ray(&rand_state, pos, x, y);     
            if (gpu_intersect_ray_model(model, face_count, pos, dir, inter))
                estimator = estimator + fabs(_dot(dir, inter.normal)) * 
                            gpu_preview_color(inter.triangle->mtl);
        }

        image[pixel_index] = (1.f / sample_count) * estimator;
    }       
    
    __device__ __host__ rgb24 _color_of_float3(const float3& color)
    {
        return {
            static_cast<unsigned char>(color.x * 255.f),
            static_cast<unsigned char>(color.y * 255.f),
            static_cast<unsigned char>(color.z * 255.f)};
    }

    std::vector<rgb24> gpu_render_outline_preview(const std::vector<gpu_face>& model, const device_camera& cam, std::size_t sample_count)
    {
        const auto width = cam.get_image_width();
        const auto height = cam.get_image_height();

        // Copy device model to device
        const auto model_size = model.size() * sizeof(gpu_face);
        gpu_face *device_model = nullptr;

        CUDA_CHECK(hipMalloc(&device_model, model_size));
        CUDA_CHECK(hipMemcpy(device_model, model.data(), model_size, hipMemcpyHostToDevice));

        // Init device image
        const auto device_image_size = width * height * sizeof(float3);
        float3 *device_image = nullptr;

        CUDA_CHECK(hipMalloc(&device_image, device_image_size));

        const auto start = std::chrono::steady_clock::now();
        preview_kernel<<<height, width>>>(device_model, model.size(), cam, sample_count, device_image);
        CUDA_CHECK(hipGetLastError());
        
        // Wait for kernel completion
        CUDA_CHECK(hipDeviceSynchronize());

        const auto end = std::chrono::steady_clock::now();

        printf("GPU computation took %d ms\n", 
            std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count());

        // Allocate host image
        std::vector<float3> output{width * height};
        std::vector<Xrender::rgb24> rgb24_output{width * height};

        //  Copy result
        CUDA_CHECK(hipMemcpy(output.data(), device_image, device_image_size, hipMemcpyDeviceToHost));

        std::transform(
            output.begin(), output.end(), rgb24_output.begin(),
            _color_of_float3);

        CUDA_CHECK(hipFree(device_model));
        CUDA_CHECK(hipFree(device_image));

        return rgb24_output;
    }

}