#include "hip/hip_runtime.h"

#include <chrono>
#include <hiprand/hiprand_kernel.h>

#include <iostream>

#include "gpu_geometric_sampler.cuh"
#include "rand_operations.cuh"
#include "cuda_exception.cuh"

namespace Xrender {

    __device__ float russian_roulette_prob(float geo_coeff, const float3& brdf_coeff)
    {
        constexpr auto threshold = 10.f / 255.f;
        constexpr auto min_prob = 0.2f;
        constexpr auto max_prob = 2.f;
        constexpr auto a = (max_prob - min_prob) / (1.f - threshold);
        constexpr auto b = max_prob - a;

        const auto estimator_norm = geo_coeff * _norm(brdf_coeff);
        const float prob = a * estimator_norm + b;
        return prob > 1.f ? 1.f : prob;
    }    
    
    __global__ void path_sampler_kernel(
        const gpu_bvh_node *bvh,
        const device_camera cam,
        const int sample_count, 
        float3 *image,
        const int width)
    {
        //  Get pixel position in image
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y;
        const int pixel_index = x + y * width;

        if (x < width)
        { 
            int sample_counter = 0;
            gpu_intersection inter;
            hiprandState rand_state;
            float3 estimator = {0.f, 0.f, 0.f};
            float3 pos;
            float3 dir;
            float3 brdf_coeff;
            float  geo_coeff;

            //  Initialize random generator
            hiprand_init(pixel_index, x*pixel_index+1, pixel_index+y, &rand_state);
            
            //  Initialize first ray
            dir = cam.sample_ray(&rand_state, pos, x, y);
            geo_coeff = dir.y;
            brdf_coeff = {1.f, 1.f, 1.f};

            while (sample_counter < sample_count)
            {
                // Russion roulette : does current ray worht the cost ?
                const float roulette_prob = russian_roulette_prob(geo_coeff, brdf_coeff);
                if (hiprand_uniform(&rand_state) <= roulette_prob)
                {
                    // keep the ray
                    geo_coeff /= roulette_prob;
                
                    //  cast a ray
                    if (gpu_intersect_ray_bvh(bvh, pos, dir, inter))
                    {
                        float3 next_dir;
                        brdf_coeff *= gpu_brdf(&rand_state, inter.mtl, inter.normal, dir, next_dir);

                        if (gpu_mtl_is_source(inter.mtl))
                        {
                            // record ray contribution
                            estimator += (geo_coeff * brdf_coeff);
                        }
                        else
                        {
                            geo_coeff *= fabs(_dot(next_dir, inter.normal));
                            pos = inter.pos;
                            dir = next_dir;
                            continue;
                        }
                    }
                }
                    
                // start a new ray
                sample_counter++;
                dir = cam.sample_ray(&rand_state, pos, x, y);
                geo_coeff = dir.y;
                brdf_coeff = {1.f, 1.f, 1.f};
            }

            image[pixel_index] = estimator * (3.f / sample_count);
        }
    }

    std::vector<float3> gpu_naive_mc(
        const std::vector<gpu_bvh_node>& tree,
        const device_camera& cam,
        const int sample_per_pixel,
        int gpu_thread_per_block)
    {
        const auto width = cam.get_image_width();
        const auto height = cam.get_image_height();
        const auto device_image_size = width * height * sizeof(float3);
        const auto device_bvh_size = tree.size() * sizeof(gpu_bvh_node);

        float3 *device_image = nullptr;
        gpu_bvh_node *device_bvh = nullptr;

        std::cout << "@GPU RENDER " << sample_per_pixel << "SPP" << std::endl;

        //  Allocate memory on device for image and model
        CUDA_CHECK(hipMalloc(&device_image, device_image_size));
        CUDA_CHECK(hipMalloc(&device_bvh, device_bvh_size));

        //  Copy model to the device
        CUDA_CHECK(hipMemcpy(device_bvh, tree.data(), device_bvh_size, hipMemcpyHostToDevice));

        //  Do the computations
        auto thread_per_block = std::min<int>(gpu_thread_per_block, width);
        const auto horizontal_block_count = static_cast<int>(ceilf((float)width / (float)thread_per_block));
        const auto start = std::chrono::steady_clock::now();
        path_sampler_kernel<<<dim3(horizontal_block_count, height), thread_per_block>>>(
            device_bvh, cam, sample_per_pixel, device_image, width);
        CUDA_CHECK(hipGetLastError());
        
        // Wait for kernel completion
        CUDA_CHECK(hipDeviceSynchronize());

        const auto end = std::chrono::steady_clock::now();

        std::cout << "GPU render took " 
                  << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count()
                  << " ms " << std::endl;

        //  Retrieve result
        std::vector<float3> result{width * height};
        CUDA_CHECK(hipMemcpy(result.data(), device_image, device_image_size, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(device_image));
        CUDA_CHECK(hipFree(device_bvh));

        return result;
    }

}