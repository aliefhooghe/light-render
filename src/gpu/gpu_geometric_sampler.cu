#include "hip/hip_runtime.h"

#include <chrono>
#include <hiprand/hiprand_kernel.h>

#include <iostream>

#include "gpu_geometric_sampler.cuh"
#include "rand_operations.cuh"
#include "cuda_exception.cuh"

namespace Xrender {

    __device__ __forceinline__ float russian_roulette_prob(float geo_coeff, const float3& brdf_coeff)
    {
        constexpr auto threshold = 10.f / 255.f;
        constexpr auto min_prob = 0.2f;
        constexpr auto max_prob = 2.f;
        constexpr auto a = (max_prob - min_prob) / (1.f - threshold);
        constexpr auto b = max_prob - a;

        const auto estimator_norm = geo_coeff * _norm(brdf_coeff);
        const float prob = a * estimator_norm + b;
        return prob > 1.f ? 1.f : prob;
    }    

    __device__ __forceinline__ float3 gpu_sample_path(
        const gpu_bvh_node *bvh,
        const float3& start_pos, const float3& start_dir, 
        hiprandState *state)
    {
        gpu_intersection inter;
        float3 pos = start_pos;
        float3 dir = start_dir;
        float3 brdf_coeff = {1.f, 1.f, 1.f};
        float geo_coeff = start_dir.y;

        for (;;)
        {
            const float prob = russian_roulette_prob(geo_coeff, brdf_coeff);

            if (hiprand_uniform(state) <= prob)
            {
                geo_coeff /= prob;
            }
            else
            {
                break;
            }

            if (gpu_intersect_ray_bvh(bvh, pos, dir, inter))
            {    
                float3 next_dir;
                brdf_coeff *= gpu_brdf(state, inter.mtl, inter.normal, dir, next_dir);

                if (gpu_mtl_is_source(inter.mtl))
                {
                    return geo_coeff * brdf_coeff;
                }
                else
                {
                    geo_coeff *= fabs(_dot(next_dir, inter.normal));
                    pos = inter.pos;
                    dir = next_dir;
                }
            }
            else
            {
                break;
            }
        }

        return {0.f, 0.f, 0.f};
    }

    __global__ void path_sampler_kernel(
        const gpu_bvh_node *bvh,
        const device_camera cam,
        const int sample_count, 
        float3 *image,
        const int width)
    {
        //  Get pixel position in image
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y;
        const int pixel_index = x + y * width;

        if (x < width) { 
            //  Initialize random generator
            hiprandState rand_state;
            hiprand_init(pixel_index, x, y, &rand_state);

            float3 pos;
            float3 dir;
            float3 estimator = {0.f, 0.f, 0.f};

            for(auto i = 0; i < sample_count; i++) {
                dir = cam.sample_ray(&rand_state, pos, x, y);
                estimator += gpu_sample_path(bvh, pos, dir, &rand_state);
            }

            image[pixel_index] = (3.f / sample_count) * estimator;
        }
    }

    std::vector<float3> gpu_naive_mc(
        const std::vector<gpu_bvh_node>& tree,
        const device_camera& cam,
        const int sample_per_pixel,
        int gpu_thread_per_block)
    {
        const auto width = cam.get_image_width();
        const auto height = cam.get_image_height();
        const auto device_image_size = width * height * sizeof(float3);
        const auto device_bvh_size = tree.size() * sizeof(gpu_bvh_node);

        float3 *device_image = nullptr;
        gpu_bvh_node *device_bvh = nullptr;

        std::cout << "@GPU RENDER " << sample_per_pixel << "SPP" << std::endl;

        //  Allocate memory on device for image and model
        CUDA_CHECK(hipMalloc(&device_image, device_image_size));
        CUDA_CHECK(hipMalloc(&device_bvh, device_bvh_size));

        //  Copy model to the device
        CUDA_CHECK(hipMemcpy(device_bvh, tree.data(), device_bvh_size, hipMemcpyHostToDevice));

        //  Do the computations
        auto thread_per_block = std::min<int>(gpu_thread_per_block, width);
        const auto horizontal_block_count = static_cast<int>(ceilf((float)width / (float)thread_per_block));
        const auto start = std::chrono::steady_clock::now();
        path_sampler_kernel<<<dim3(horizontal_block_count, height), thread_per_block>>>(
            device_bvh, cam, sample_per_pixel, device_image, width);
        CUDA_CHECK(hipGetLastError());
        
        // Wait for kernel completion
        CUDA_CHECK(hipDeviceSynchronize());

        const auto end = std::chrono::steady_clock::now();

        std::cout << "GPU render took " 
                  << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count()
                  << " ms " << std::endl;

        //  Retrieve result
        std::vector<float3> result{width * height};
        CUDA_CHECK(hipMemcpy(result.data(), device_image, device_image_size, hipMemcpyDeviceToHost));

        CUDA_CHECK(hipFree(device_image));
        CUDA_CHECK(hipFree(device_bvh));

        return result;
    }

}