#include "hip/hip_runtime.h"

#include <algorithm>
#include <chrono>
#include <iostream>

#include "gpu/model/bvh_tree_traversal.cuh"
#include "gpu/model/material_brdf.cuh"
#include "gpu/utils/cuda_exception.cuh"
#include "gpu/utils/curand_helper.cuh"
#include "gpu/utils/image_grid_dim.cuh"

#include "naive_mc_renderer.cuh"

namespace Xrender {

    __global__ void render_develop_to_surface_kernel(
        const float3 *sensor, hipSurfaceObject_t surface, float factor, const int width)
    {
        //  Get pixel position in image
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y;

        if (x < width) {
            const int pixel_index = x + y * width;

            const auto rgb_value = sensor[pixel_index] * factor;
            const float4 rgba_value = {
                rgb_value.x,
                rgb_value.y,
                rgb_value.z,
                1.f
            };

            surf2Dwrite(rgba_value, surface, x * sizeof(float4), y);
        }
    }

    __device__ float russian_roulette_prob(const float3& bounce_coeff)
    {
        constexpr auto factor = 1.f;
        const auto refl = fmaxf(bounce_coeff.x, fmaxf(bounce_coeff.y, bounce_coeff.z));
        return fminf(factor * refl, 1.f);
    }

    __global__ void path_sampler_kernel(
        const bvh_node *bvh,
        const camera cam,
        const int sample_count,
        hiprandState_t *rand_pool,
        float3 *sensor)
    {
        const auto width = cam.get_image_width();

        //  Get pixel position in image
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y;
        const int pixel_index = x + y * width;

        if (x < width)
        {
            int sample_counter = 0;
            intersection inter;
            auto rand_state = rand_pool[pixel_index];
            float3 estimator = {0.f, 0.f, 0.f};
            float3 pos;
            float3 dir;
            float3 brdf_coeff;
            float russian_roulette_factor = 1.f;

            //  Initialize first ray
            int bounce = 0;
            dir = cam.sample_ray(&rand_state, pos, x, y);
            russian_roulette_factor = 1.f;
            brdf_coeff = {1.f, 1.f, 1.f};

            while (sample_counter < sample_count)
            {
                //  cast a ray
                if (intersect_ray_bvh(bvh, pos, dir, inter))
                {
                    float3 next_dir;
                    const float3 bounce_coeff = sample_brdf(&rand_state, inter, inter.normal, dir, next_dir);
                    brdf_coeff *= bounce_coeff;

                    if (gpu_mtl_is_source(inter.mtl))
                    {
                        // record ray contribution
                        estimator += (russian_roulette_factor * brdf_coeff);
                    }
                    else
                    {
                        // Russion roulette : does current ray worht the cost ?
                        const float roulette_prob = russian_roulette_prob(bounce_coeff);

                        if (hiprand_uniform(&rand_state) < roulette_prob)
                        {
                            // continue the ray
                            russian_roulette_factor /= roulette_prob;
                            pos = inter.pos;
                            dir = next_dir;
                            bounce++;
                            continue;
                        }
                    }
                }

                // start a new ray
                sample_counter++;
                bounce=0;
                dir = cam.sample_ray(&rand_state, pos, x, y);
                russian_roulette_factor = 1.f;
                brdf_coeff = {1.f, 1.f, 1.f};
            }

            sensor[pixel_index] += estimator;
            rand_pool[pixel_index] = rand_state;
        }
    }

    naive_mc_renderer::naive_mc_renderer(const bvh_node *device_tree, camera& cam)
    :   gpu_renderer{cam},
        _device_tree{device_tree}
    {
    }

    __host__ void naive_mc_renderer::_call_integrate_kernel(std::size_t sample_count, hiprandState_t *rand_pool, float3 *sensor)
    {
        path_sampler_kernel<<<_image_grid_dim(), _image_thread_per_block()>>>(
            _device_tree, _camera, sample_count, rand_pool, sensor);
    }

    __host__ void naive_mc_renderer::_call_develop_to_texture_kernel(const float3 *sensor, hipSurfaceObject_t texture)
    {
        const auto factor = 1.f / get_total_sample_count();
        render_develop_to_surface_kernel<<<_image_grid_dim(), _image_thread_per_block()>>>(
            sensor, texture, factor, _camera.get_image_width());
    }
}