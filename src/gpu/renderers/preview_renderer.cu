#include "hip/hip_runtime.h"

#include <algorithm>
#include <chrono>
#include <iostream>

#include "gpu/utils/cuda_exception.cuh"
#include "gpu/utils/curand_helper.cuh"
#include "gpu/utils/image_grid_dim.cuh"
#include "gpu/model/bvh_tree_traversal.cuh"

#include "preview_renderer.cuh"

namespace Xrender
{
    __global__
    __launch_bounds__(preview_renderer::max_thread_per_block)
    void preview_integrate_kernel(
        const bvh_node *tree, int tree_size,
        const face *geometry,
        const material *mtl_bank,
        const camera cam,
        hiprandState_t *rand_pool,
        const int sample_count,
        float3 *image)
    {
        //  Get pixel position in ima
        const auto width = cam.get_image_width();

        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y;
        const int pixel_index = x + y * width;

        //  Initialize random generator
        if (x < width)
        {
            auto rand_state = rand_pool[pixel_index];

            float3 pos;
            float3 dir;
            intersection inter;
            float3 estimator = {0.f, 0.f, 0.f};

            for (auto i = 0; i < sample_count; i++)
            {
                dir = cam.sample_ray(&rand_state, pos, x, y);

                int best_geo_index;
                if (intersect_ray_bvh(tree, tree_size, geometry, pos, dir, inter, best_geo_index))
                {
                    const auto mtl_index = geometry[best_geo_index].mtl;
                    const auto normal = interpolate_normal(dir, inter.uv, geometry[best_geo_index].geo.normals);
                    const auto mtl = mtl_bank[mtl_index];
                    estimator += gpu_preview_color(mtl) * -dot(dir, normal);
                }
                else
                {
                    estimator += float3{0.f, 0.f, 1.f};
                }
            }

            image[pixel_index] += estimator;
            rand_pool[pixel_index] = rand_state;
        }
    }

    preview_renderer::preview_renderer(
        const bvh_node *device_tree, int tree_size,
        const face *device_model,
        const material *device_mtl_bank)
        : _device_tree{device_tree},
          _tree_size{tree_size},
          _device_model{device_model},
          _device_mtl_bank{device_mtl_bank}
    {
    }

    void preview_renderer::call_integrate_kernel(
        const camera &cam, hiprandState_t *rand_pool, std::size_t sample_count, float3 *sensor)
    {
        unsigned int thread_per_block = max_thread_per_block;
        const auto grid_dim = image_grid_dim(
            cam.get_image_width(), cam.get_image_height(), thread_per_block);

        preview_integrate_kernel<<<grid_dim, thread_per_block, 0>>>(
            _device_tree, _tree_size, _device_model, _device_mtl_bank, cam, rand_pool, sample_count, sensor);
    }
}