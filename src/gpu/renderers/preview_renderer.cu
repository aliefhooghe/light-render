#include "hip/hip_runtime.h"

#include <algorithm>
#include <chrono>
#include <iostream>

#include "gpu/utils/cuda_exception.cuh"
#include "gpu/utils/curand_helper.cuh"
#include "gpu/utils/image_grid_dim.cuh"
#include "gpu/model/bvh_tree_traversal.cuh"

#include "preview_renderer.cuh"

namespace Xrender
{
    __global__ void preview_integrate_kernel(
        const bvh_node *tree,
        const face *geometry,
        const material *mtl_bank,
        const camera cam,
        hiprandState_t *rand_pool,
        const int sample_count,
        float3 *image)
    {
        //  Get pixel position in ima
        const auto width = cam.get_image_width();

        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y;
        const int pixel_index = x + y * width;

        //  Initialize random generator
        if (x < width)
        {
            auto rand_state = rand_pool[pixel_index];

            float3 pos;
            float3 dir;
            intersection inter;
            float3 estimator = {0.f, 0.f, 0.f};

            for (auto i = 0; i < sample_count; i++)
            {
                dir = cam.sample_ray(&rand_state, pos, x, y);

                int best_geo_index;
                if (intersect_ray_bvh(tree, geometry, pos, dir, inter, best_geo_index))
                {
                    const auto mtl_index = geometry[best_geo_index].mtl;
                    const auto normal = interpolate_normal(dir, inter.uv, geometry[best_geo_index].geo.normals);
                    const auto mtl = mtl_bank[mtl_index];
                    estimator += gpu_preview_color(mtl) * -dot(dir, normal);
                }
                else
                {
                    estimator += float3{0.f, 0.f, 1.f};
                }
            }

            image[pixel_index] += estimator;
            rand_pool[pixel_index] = rand_state;
        }
    }

    preview_renderer::preview_renderer(
        const bvh_node *device_tree,
        const face *device_model,
        const material *device_mtl_bank,
        std::size_t thread_per_block)
        : _device_tree{device_tree},
          _device_model{device_model},
          _device_mtl_bank{device_mtl_bank},
          _thread_per_block{thread_per_block}
    {
    }

    void preview_renderer::call_integrate_kernel(
        const camera &cam, hiprandState_t *rand_pool, std::size_t sample_count, float3 *sensor)
    {
        unsigned int thread_per_block = _thread_per_block;
        const auto grid_dim = image_grid_dim(
            cam.get_image_width(), cam.get_image_height(), thread_per_block);

        preview_integrate_kernel<<<grid_dim, thread_per_block>>>(
            _device_tree, _device_model, _device_mtl_bank, cam, rand_pool, sample_count, sensor);
    }
}