#include "hip/hip_runtime.h"

#include <stdexcept>
#include <iostream>
#include <iomanip>
#include <cmath>

#include "gpu/utils/cuda_exception.cuh"
#include "gpu/utils/curand_pool.cuh"

#include "renderer_manager.cuh"

namespace Xrender {

    renderer_manager::renderer_manager(
        const camera& cam,
        std::unique_ptr<abstract_renderer>&& renderer)
    :   _renderer{std::move(renderer)},
        _camera{cam}
    {
        if (!_renderer)
            throw std::invalid_argument("Renderer manager : renderer is null");

        const auto width = _camera.get_image_width();
        const auto height = _camera.get_image_height();

        // Init device sensor
        CUDA_CHECK(hipMalloc(&_device_sensor, width * height * sizeof(float3)));
        reset();

        // Init random generator pool
        _rand_pool = create_curand_pool(width * height);
    }

    renderer_manager::renderer_manager(renderer_manager&& other) noexcept
    :   _renderer{std::move(other._renderer)},
        _status{other._status},
        _camera{other._camera},
        _rand_pool{other._rand_pool},
        _device_sensor{other._device_sensor}
    {
        // Avoir multiple desalocation
        other._rand_pool = nullptr;
        other._device_sensor = nullptr;
    }

    renderer_manager::~renderer_manager()
    {
        if (_rand_pool) CUDA_WARNING(hipFree(_rand_pool));
        if (_device_sensor) CUDA_WARNING(hipFree(_device_sensor));
    }

    void renderer_manager::reset()
    {
        const auto width = _camera.get_image_width();
        const auto height = _camera.get_image_height();
        CUDA_CHECK(hipMemset(_device_sensor, 0u, width * height * sizeof(float3)));
        _status.total_integrated_sample = 0u;
    }

    void renderer_manager::integrate_for(const std::chrono::milliseconds& max_duration)
    {
        using namespace std::chrono;

        // estimate sample count to run
        _status.last_sample_count = std::max<std::size_t>(1u, static_cast<std::size_t>(
            _status.spp_per_second * static_cast<float>(max_duration.count()) / 1000.f));

        auto start_time = steady_clock::now();
        _render_integrate_step(_status.last_sample_count);
        auto end_time = steady_clock::now();
        auto duration = end_time - start_time;

        // Update sample per step to follow the interval requirement

        const auto recorded_speed = static_cast<float>(_status.last_sample_count ) / static_cast<float>(duration.count());
        const auto recorded_speed_spp_per_sec = recorded_speed * 1E9f;

        std::cout
            << "\r[ Integrated " << _status.last_sample_count  << " more samples ] - [ "
            << std::fixed << recorded_speed_spp_per_sec << " spp/sec - total = " << _status.total_integrated_sample << " spp ]" << std::flush;

        _status.spp_per_second = recorded_speed_spp_per_sec;
    }

    void renderer_manager::_render_integrate_step(std::size_t sample_count)
    {
        _renderer->call_integrate_kernel(_camera, _rand_pool, sample_count, _device_sensor);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        _status.total_integrated_sample += sample_count;
    }
}