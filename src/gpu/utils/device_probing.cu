
#include <hip/hip_runtime.h>
#include <iostream>

#include "cuda_exception.cuh"

namespace Xrender
{
    bool select_openGL_cuda_device()
    {
        int device_count;

        CUDA_CHECK(hipGetDeviceCount(&device_count));

        for (auto id = 0; id < device_count; ++id)
        {
            int compute_mode;
            int is_integrated;
            CUDA_CHECK(hipDeviceGetAttribute(&compute_mode, hipDeviceAttributeComputeMode, id));
            CUDA_CHECK(hipDeviceGetAttribute(&is_integrated, hipDeviceAttributeIntegrated, id));

            if (compute_mode != hipComputeModeProhibited)
            {
                hipDeviceProp_t device_prop;
                CUDA_CHECK(hipGetDeviceProperties(&device_prop, id));
                CUDA_CHECK(hipSetDevice(id));
                std::cout << "Found cuda capable device : " << device_prop.name << std::endl;
                return true;
            }
        }

        return false;
    }
}